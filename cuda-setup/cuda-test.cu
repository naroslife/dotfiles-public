#include "hip/hip_runtime.h"
/**
 * CUDA Test Program
 *
 * This program tests basic CUDA functionality including:
 * - Device detection and properties
 * - Memory allocation and transfer
 * - Kernel execution
 * - Error handling
 */

#include <stdio.h>
#include <hip/hip_runtime.h>

#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            fprintf(stderr, "CUDA Error at %s:%d - %s\n", __FILE__, __LINE__, \
                    hipGetErrorString(error)); \
            return 1; \
        } \
    } while(0)

// Simple kernel that adds two numbers
__global__ void add_kernel(int *a, int *b, int *c) {
    *c = *a + *b;
}

// Kernel that demonstrates thread parallelism
__global__ void hello_kernel(int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        printf("  Thread %d: Hello from GPU!\n", idx);
    }
}

// Vector addition kernel
__global__ void vector_add_kernel(const float *a, const float *b, float *c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

void print_separator() {
    printf("\n========================================\n");
}

int test_device_info() {
    print_separator();
    printf("TEST 1: Device Information\n");
    print_separator();

    int device_count;
    CUDA_CHECK(hipGetDeviceCount(&device_count));

    printf("Found %d CUDA device(s)\n\n", device_count);

    if (device_count == 0) {
        printf("No CUDA devices found!\n");
        return 1;
    }

    for (int i = 0; i < device_count; i++) {
        hipDeviceProp_t prop;
        CUDA_CHECK(hipGetDeviceProperties(&prop, i));

        printf("Device %d: %s\n", i, prop.name);
        printf("  Compute Capability: %d.%d\n", prop.major, prop.minor);
        printf("  Total Global Memory: %.2f GB\n", prop.totalGlobalMem / 1e9);
        printf("  Multiprocessors: %d\n", prop.multiProcessorCount);
        printf("  Max Threads per Block: %d\n", prop.maxThreadsPerBlock);
        printf("  Max Threads per MP: %d\n", prop.maxThreadsPerMultiProcessor);
        printf("  Warp Size: %d\n", prop.warpSize);
        printf("  Memory Clock Rate: %.2f GHz\n", prop.memoryClockRate / 1e6);
        printf("  Memory Bus Width: %d-bit\n", prop.memoryBusWidth);
        printf("  L2 Cache Size: %.2f MB\n", prop.l2CacheSize / 1e6);

        if (i < device_count - 1) {
            printf("\n");
        }
    }

    printf("\n✓ Device information test passed\n");
    return 0;
}

int test_simple_kernel() {
    print_separator();
    printf("TEST 2: Simple Kernel Execution\n");
    print_separator();

    int a = 5, b = 7;
    int *d_a, *d_b, *d_c;
    int c;

    // Allocate device memory
    printf("Allocating device memory...\n");
    CUDA_CHECK(hipMalloc(&d_a, sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_b, sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_c, sizeof(int)));

    // Copy data to device
    printf("Copying data to device...\n");
    CUDA_CHECK(hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice));

    // Launch kernel
    printf("Launching kernel: %d + %d = ?\n", a, b);
    add_kernel<<<1, 1>>>(d_a, d_b, d_c);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy result back
    printf("Copying result back to host...\n");
    CUDA_CHECK(hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost));

    printf("Result: %d + %d = %d\n", a, b, c);

    // Cleanup
    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_c));

    if (c == a + b) {
        printf("\n✓ Simple kernel test passed\n");
        return 0;
    } else {
        printf("\n✗ Simple kernel test FAILED (expected %d, got %d)\n", a + b, c);
        return 1;
    }
}

int test_parallel_execution() {
    print_separator();
    printf("TEST 3: Parallel Thread Execution\n");
    print_separator();

    int n = 8;
    printf("Launching %d threads to print messages:\n\n", n);

    hello_kernel<<<1, n>>>(n);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    printf("\n✓ Parallel execution test passed\n");
    return 0;
}

int test_vector_addition() {
    print_separator();
    printf("TEST 4: Vector Addition\n");
    print_separator();

    const int n = 1024;
    const size_t bytes = n * sizeof(float);

    // Allocate host memory
    printf("Allocating host memory for %d elements...\n", n);
    float *h_a = (float*)malloc(bytes);
    float *h_b = (float*)malloc(bytes);
    float *h_c = (float*)malloc(bytes);

    // Initialize host arrays
    printf("Initializing arrays...\n");
    for (int i = 0; i < n; i++) {
        h_a[i] = (float)i;
        h_b[i] = (float)i * 2.0f;
    }

    // Allocate device memory
    printf("Allocating device memory...\n");
    float *d_a, *d_b, *d_c;
    CUDA_CHECK(hipMalloc(&d_a, bytes));
    CUDA_CHECK(hipMalloc(&d_b, bytes));
    CUDA_CHECK(hipMalloc(&d_c, bytes));

    // Copy data to device
    printf("Copying data to device...\n");
    CUDA_CHECK(hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice));

    // Launch kernel
    printf("Launching vector addition kernel...\n");
    int threads_per_block = 256;
    int blocks = (n + threads_per_block - 1) / threads_per_block;
    printf("  Grid: %d blocks x %d threads = %d total threads\n",
           blocks, threads_per_block, blocks * threads_per_block);

    vector_add_kernel<<<blocks, threads_per_block>>>(d_a, d_b, d_c, n);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy result back
    printf("Copying result back to host...\n");
    CUDA_CHECK(hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost));

    // Verify results
    printf("Verifying results...\n");
    int errors = 0;
    for (int i = 0; i < n; i++) {
        float expected = h_a[i] + h_b[i];
        if (h_c[i] != expected) {
            if (errors < 5) {
                printf("  Error at index %d: expected %.2f, got %.2f\n",
                       i, expected, h_c[i]);
            }
            errors++;
        }
    }

    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);

    if (errors == 0) {
        printf("\n✓ Vector addition test passed (%d elements)\n", n);
        return 0;
    } else {
        printf("\n✗ Vector addition test FAILED (%d errors)\n", errors);
        return 1;
    }
}

int main() {
    printf("\n");
    printf("==========================================\n");
    printf("       CUDA Functionality Test Suite     \n");
    printf("==========================================\n");

    // Get CUDA runtime version
    int runtime_version;
    hipRuntimeGetVersion(&runtime_version);
    printf("\nCUDA Runtime Version: %d.%d\n",
           runtime_version / 1000, (runtime_version % 100) / 10);

    int driver_version;
    hipDriverGetVersion(&driver_version);
    printf("CUDA Driver Version: %d.%d\n",
           driver_version / 1000, (driver_version % 100) / 10);

    // Run tests
    int failed = 0;

    if (test_device_info() != 0) failed++;
    if (test_simple_kernel() != 0) failed++;
    if (test_parallel_execution() != 0) failed++;
    if (test_vector_addition() != 0) failed++;

    // Summary
    print_separator();
    printf("TEST SUMMARY\n");
    print_separator();

    if (failed == 0) {
        printf("\n✓ All tests passed!\n");
        printf("\nYour CUDA installation is working correctly.\n\n");
        return 0;
    } else {
        printf("\n✗ %d test(s) failed!\n", failed);
        printf("\nThere are issues with your CUDA installation.\n\n");
        return 1;
    }
}